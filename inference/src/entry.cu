#include "hip/hip_runtime.h"
#include <pybind11/pybind11.h>
#include <hip/hip_runtime.h>

#include "utils.cuh"
#include "trait.cuh"
#include "model/model.cuh"
#include "model/medusa.cuh"
#include "model/eagle.cuh"

#define DTYPE_SWITCH(COND, ...)               \
  [&] {                                      \
    if (COND == 0) {                              \
      using elem_type = __half;     \
      return __VA_ARGS__();                  \
    } else {                                 \
      using elem_type = __hip_bfloat16; \
      return __VA_ARGS__();                  \
    }                                        \
  }()

Model* model;

void init_base_model(
    int64_t memory_limit,
    std::uintptr_t memory_pool,
    int vocab_size,
    int num_hidden_layers,
    int hidden_size,
    int intermediate_size,
    int num_attention_heads,
    int num_key_value_heads,
    int head_dim,
    float rms_norm_eps,
    int torch_dtype,
    int chunk_length,
    bool use_kernel
) {
    init_resources();

    DTYPE_SWITCH(torch_dtype, [&] {
        model = new ModelImpl<elem_type>(
            memory_limit,
            reinterpret_cast<void*>(memory_pool),
            vocab_size,
            num_hidden_layers,
            hidden_size,
            intermediate_size,
            num_attention_heads,
            num_key_value_heads,
            head_dim,
            rms_norm_eps,
            chunk_length,
            use_kernel
        );
    });

}

void init_medusa_model(
    int num_heads,
    int num_layers,
    int topk_per_head,
    int tree_size,
    std::uintptr_t tree_indices,
    std::uintptr_t draft_position_ids,
    int torch_dtype
) {
    DTYPE_SWITCH(torch_dtype, [&] {
        model = new MedusaImpl<elem_type>(
            (ModelImpl<elem_type>*)model,
            num_heads,
            num_layers,
            topk_per_head,
            tree_size,
            reinterpret_cast<int32_t*>(tree_indices),
            reinterpret_cast<int32_t*>(draft_position_ids)
        );
    });
}

void init_eagle_model(
    int num_layers,
    int intermediate_size,
    int num_attention_heads,
    int num_key_value_heads,
    int head_dim,
    int num_iter,
    int topk_per_iter,
    int tree_size,
    int V,
    int torch_dtype
) {
    DTYPE_SWITCH(torch_dtype, [&] {
        model = new EagleImpl<elem_type>(
            (ModelImpl<elem_type>*)model,
            num_layers,
            intermediate_size,
            num_attention_heads,
            num_key_value_heads,
            head_dim,
            num_iter,
            topk_per_iter,
            tree_size,
            V
        );
    });
}

int init_storage() {
    return model->init_storage();
}

void load_model(std::string name, std::uintptr_t param) {
    model->load_to_storage(name, reinterpret_cast<void*>(param));
}

void prefill(int input_length, int history_length, std::uintptr_t input, std::uintptr_t position_ids, std::uintptr_t output) {
    model->prefill(input_length, history_length, reinterpret_cast<int32_t*>(input), reinterpret_cast<int32_t*>(position_ids), (void*)(output));
}

void decode(int input_length, int padded_length, std::uintptr_t input, std::uintptr_t position_ids, std::uintptr_t cache_length, std::uintptr_t mask_2d, std::uintptr_t output, bool cuda_graph) {
    if (cuda_graph) {
        if (graphCreated_padding_length != padded_length || graphCreated_input_length != input_length) {
            if (graphExec != nullptr) {
                hipGraphExecDestroy(graphExec);
                graphExec = nullptr;
            }
            if (graph != nullptr) {
                hipGraphDestroy(graph);
                graph = nullptr;
            }
            hipStreamBeginCapture(calc_stream.stream, hipStreamCaptureModeGlobal);
            model->decode(input_length, padded_length, reinterpret_cast<int32_t*>(input), reinterpret_cast<int32_t*>(position_ids), reinterpret_cast<int32_t*>(cache_length), reinterpret_cast<uint64_t*>(mask_2d), reinterpret_cast<void*>(output));
            hipStreamEndCapture(calc_stream.stream, &graph);
            hipGraphInstantiate(&graphExec, graph, nullptr, nullptr, 0);
            graphCreated_padding_length = padded_length;
            graphCreated_input_length = input_length;
        }
        hipGraphLaunch(graphExec, calc_stream.stream);
    } else {
        model->decode(input_length, padded_length, reinterpret_cast<int32_t*>(input), reinterpret_cast<int32_t*>(position_ids), reinterpret_cast<int32_t*>(cache_length), reinterpret_cast<uint64_t*>(mask_2d), reinterpret_cast<void*>(output));
    }
}

void draft(std::uintptr_t tree_draft_ids, std::uintptr_t tree_position_ids, std::uintptr_t cache_length, std::uintptr_t attn_mask, std::uintptr_t tree_parent) {
    model->draft(reinterpret_cast<int32_t*>(tree_draft_ids), reinterpret_cast<int32_t*>(tree_position_ids), reinterpret_cast<int32_t*>(cache_length), reinterpret_cast<uint64_t*>(attn_mask), reinterpret_cast<int32_t*>(tree_parent));
}

int verify_and_fix(int num_tokens, std::uintptr_t pred, std::uintptr_t gt, std::uintptr_t position_ids, std::uintptr_t cache_length, std::uintptr_t attn_mask, std::uintptr_t tree_parent) {
    return model->verify(num_tokens, reinterpret_cast<int32_t*>(pred), reinterpret_cast<int32_t*>(gt), reinterpret_cast<int32_t*>(position_ids), reinterpret_cast<int32_t*>(cache_length), reinterpret_cast<uint64_t*>(attn_mask), reinterpret_cast<int32_t*>(tree_parent));
}

PYBIND11_MODULE(C, m) {
    m.def("init_base_model", &init_base_model, "Init base model");
    m.def("init_medusa_model", &init_medusa_model, "Init medusa model");
    m.def("init_eagle_model", &init_eagle_model, "Init eagle model");
    m.def("init_storage", &init_storage, "Init storage");
    m.def("load_model", &load_model, "Load model");
    m.def("prefill", &prefill, "Prefill");
    m.def("decode", &decode, "Decode");
    m.def("draft", &draft, "Draft");
    m.def("verify_and_fix", &verify_and_fix, "Verify and fix");
} 