#include "utils.cuh"

bool initialized = false;

Stream calc_stream;

int graphCreated_padding_length = -1;
int graphCreated_input_length = -1;
hipGraph_t graph;
hipGraphExec_t graphExec;

void init_resources() {
  if (initialized) return;
  cudaCheck(hipStreamCreate(&calc_stream.stream));
  cublasCheck(hipblasCreate(&calc_stream.cublas_handle));
  cublasCheck(hipblasSetStream(calc_stream.cublas_handle, calc_stream.stream));
  initialized = true;
}
